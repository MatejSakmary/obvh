#include "CubWrapper.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#define MEASURE_EXECUTION_TIMES

// In and out buffers may be swaped
// Original data is not kept
template <typename T> float DeviceSort(unsigned int numberOfElements, T* keysIn, T* keysOut,
                 unsigned int* valuesIn, unsigned int* valuesOut)
{
    //hipcub::DoubleBuffer<T> keysBuffer(*keysIn, *keysOut);
    //hipcub::DoubleBuffer<unsigned int> valuesBuffer(*valuesIn, *valuesOut);

    // Check how much temporary memory will be required
    void* tempStorage = nullptr;
    size_t storageSize = 0;
    hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize,
        keysIn, keysOut, valuesIn, valuesOut, numberOfElements);
    //hipcub::DeviceRadixSort::SortKeys(tempStorage, storageSize, keysBuffer, numberOfElements);

    // Allocate temporary memory
    hipMalloc(&tempStorage, storageSize);

    float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
#endif

    // Sort
    hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize,
        keysIn, keysOut, valuesIn, valuesOut, numberOfElements);

#ifdef MEASURE_EXECUTION_TIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
#endif

    // Free temporary memory
    hipFree(tempStorage);
    return elapsedTime;
}

float DeviceSort(unsigned int numberOfElements, unsigned int* keysIn, unsigned int* keysOut,
    unsigned int* valuesIn, unsigned int* valuesOut)
{
    return DeviceSort<unsigned int>(numberOfElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(unsigned int numberOfElements, unsigned long long int* keysIn, unsigned long long int* keysOut,
    unsigned int* valuesIn, unsigned int* valuesOut)
{
    return DeviceSort<unsigned long long int>(numberOfElements, keysIn, keysOut, valuesIn, valuesOut);
}
